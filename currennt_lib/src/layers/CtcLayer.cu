#include "hip/hip_runtime.h"
#include "CtcLayer.hpp"

namespace internal
{
namespace
{
    class Log
    {
        real_t expVal,logVal;
    public:
        static real_t expMax,expMin,expLimit,logZero,logInfinity;
        static real_t safe_exp(real_t x)
        {
            if (x==logZero)
                return(0);
            if (x>=expLimit)
                return(expMax);
            return(std::exp(x));
        }
        static real_t safe_log(real_t x)
        {
            if (x<expMin)
                return(logZero);
            return(std::log(x));
        }
        static real_t log_add(real_t x,real_t y)
        {
            if (x==logZero)
                return(y);
            if (y==logZero)
                return(x);
            if (x<y)
                thrust::swap(x,y);
            return(x+std::log(1.0+safe_exp(y-x)));
        }
        static real_t log_subtract(real_t x,real_t y)
        {
            if (y==logZero)
                return(x);
            if (y>=x)
                return(logZero);
            return(x+std::log(1.0-safe_exp(y-x)));
        }
        static real_t log_multiply(real_t x,real_t y)
        {
            if (x==logZero || y==logZero)
                return(logZero);
            return(x+y);
        }
        static real_t log_divide(real_t x,real_t y)
        {
            if (x==logZero)
                return(logZero);
            if (y==logZero)
                return(logInfinity);
            return(x-y);
        }
        Log(real_t v=0,bool logScale=false):expVal(logScale?-1:v),logVal(logScale?v:safe_log(v)){}
        Log &operator =(const Log &l)
        {
            logVal=l.logVal;
            expVal=l.expVal;
            return(*this);
        }
        Log &operator +=(const Log &l)
        {
            logVal=log_add(logVal,l.logVal);
            expVal=-1;
            return(*this);
        }
        Log &operator -=(const Log &l)
        {
            logVal=log_subtract(logVal,l.logVal);
            expVal=-1;
            return(*this);
        }
        Log &operator *=(const Log &l)
        {
            logVal=log_multiply(logVal,l.logVal);
            expVal=-1;
            return(*this);
        }
        Log &operator /=(const Log &l)
        {
            logVal=log_divide(logVal,l.logVal);
            expVal=-1;
            return(*this);
        }
        real_t exp()
        {
            if (expVal<0)
                expVal=safe_exp(logVal);
            return(expVal);
        }
        real_t log() const
        {
            return(logVal);
        }
    };
    Log operator +(const Log &log1,const Log &log2)
    {
        return(Log(Log::log_add(log1.log(),log2.log()),true));
    }
    Log operator -(const Log &log1,const Log &log2)
    {
        return(Log(Log::log_subtract(log1.log(),log2.log()),true));
    }
    Log operator *(const Log &log1,const Log &log2)
    {
        return(Log(Log::log_multiply(log1.log(),log2.log()),true));
    }
    Log operator /(const Log &log1,const Log &log2)
    {
        return(Log(Log::log_divide(log1.log(),log2.log()),true));
    }
    bool operator <(const Log &log1,const Log &log2)
    {
        return(log1.log()<log2.log());
    }
    bool operator >(const Log &log1,const Log &log2)
    {
        return(log1.log()>log2.log());
    }
    bool operator <=(const Log &log1,const Log &log2)
    {
        return(log1.log()<=log2.log());
    }
    bool operator >=(const Log &log1,const Log &log2)
    {
        return(log1.log()>=log2.log());
    }
    bool operator ==(const Log &log1,const Log &log2)
    {
        return(log1.log()==log2.log());
    }
    bool operator !=(const Log &log1,const Log &log2)
    {
        return(log1.log()!=log2.log());
    }
    std::istream &operator >>(std::istream &in,Log &l)
    {
        real_t d;
        in>>d;
        l=Log(d,true);
        return(in);
    }
    std::ostream &operator <<(std::ostream &out,const Log &l)
    {
        out<<l.log();
        return(out);
    }
    real_t Log::expMax=std::numeric_limits<real_t>::max();
    real_t Log::expMin=std::numeric_limits<real_t>::min();
    real_t Log::expLimit=std::log(expMax);
    real_t Log::logInfinity=1e38;
    real_t Log::logZero=-Log::logInfinity;
}
}

namespace layers
{

    template <typename TDevice> CtcLayer <TDevice>::CtcLayer(const helpers::JsonValue &layerChild,TrainableLayer <TDevice> &precedingLayer)
        :PostOutputLayer <TDevice>(layerChild,precedingLayer)
    {
    }

    template <typename TDevice> CtcLayer <TDevice>::~CtcLayer()
    {
    }

    template <typename TDevice> real_t CtcLayer <TDevice>::calculateError()
    {
        output.resize(this->_actualOutputs().size());
        thrust::copy(this->_actualOutputs().begin(),this->_actualOutputs().end(),output.begin());
        outputErrors.resize(this->_outputErrors().size());
        thrust::fill(outputErrors.begin(),outputErrors.end(),0);
        real_t error=0;
        for (int i=0;i<realSeqNum;i++)
        {
            error+=calculateError(i);
            targetLabel[i].clear();
        }
        delete[] targetLabel;
        return(error);
    }

    template <typename TDevice> real_t CtcLayer <TDevice>::calculateError(int id)
    {

        totalTime=targetLabel[id].size();
        targetLabel[id].erase(thrust::find(targetLabel[id].begin(),targetLabel[id].end(),-1),targetLabel[id].end());
        const int_vector &targetLabelSeq=targetLabel[id];
        typedef thrust::host_vector <internal::Log> ctc_vector;
        int blank=this->size()-1;
        totalSegments=targetLabelSeq.size()*2+1;

        //calculate the forward variables
        ctc_vector *forwardVariables=new ctc_vector[totalTime];
        forwardVariables[0].resize(totalSegments);
        forwardVariables[0][0]=activation(id,0,blank);
        if (totalSegments>1)
            forwardVariables[0][1]=activation(id,0,targetLabelSeq[0]);
        for (int t=1;t<totalTime;t++)
        {
            const ctc_vector &oldFvars=forwardVariables[t-1];
            ctc_vector &fvars=forwardVariables[t];
            fvars.resize(totalSegments);
            thrust::pair <int,int> bound=segment_range(t);
            for (int s=bound.first;s<bound.second;s++)
            {
                internal::Log fv;
                if (s&1)
                {
                    int labelIndex=s/2;
                    int labelNum=targetLabelSeq[labelIndex];
                    fv=oldFvars[s]+oldFvars[s-1];
                    if (s>1 && labelNum!=targetLabelSeq[labelIndex-1])
                        fv+=oldFvars[s-2];
                    fv*=activation(id,t,labelNum);
                }
                else
                {
                    fv=oldFvars[s];
                    if (s)
                        fv+=oldFvars[s-1];
                    fv*=activation(id,t,blank);
                }
                fvars[s]=fv;
            }
        }

        //calculate the backward vairables
        ctc_vector *backwardVariables=new ctc_vector[totalTime];
        ctc_vector &lastBvs=backwardVariables[totalTime-1];
        lastBvs.resize(totalSegments);
        lastBvs.back()=1;
        if (totalSegments>1)
           lastBvs[lastBvs.size()-2]=1;
        for (int t=totalTime-2;t>=0;t--)
        {
            const ctc_vector &oldBvars=backwardVariables[t+1];
            ctc_vector &bvars=backwardVariables[t];
            bvars.resize(totalSegments);
            thrust::pair <int,int> bound=segment_range(t);
            for (int s=bound.first;s<bound.second;s++)
            {
                internal::Log bv;
                if (s&1)
                {
                    int labelIndex=s/2;
                    int labelNum=targetLabelSeq[labelIndex];
                    bv=oldBvars[s]*activation(id,t+1,labelNum)+oldBvars[s+1]*activation(id,t+1,blank);
                    if (s<totalSegments-2)
                    {
                        int nextLabelNum=targetLabelSeq[labelIndex+1];
                        if (labelNum!=nextLabelNum)
                            bv+=oldBvars[s+2]*activation(id,t+1,nextLabelNum);
                    }
                }
                else
                {
                    bv=oldBvars[s]*activation(id,t+1,blank);
                    if (s<totalSegments-1)
                        bv+=oldBvars[s+1]*activation(id,t+1,targetLabelSeq[s/2]);
                }
                bvars[s]=bv;
            }
        }

        //calcuate the errors
        const ctc_vector &lastFvs=forwardVariables[totalTime-1];
        internal::Log logProb=lastFvs.back();
        if (totalSegments>1)
            logProb+=lastFvs[lastFvs.size()-2];

        if (logProb==0.0)
        {
            for (int i=0;i<totalTime;i++)
            {
                forwardVariables[i].clear();
                backwardVariables[i].clear();
            }
            delete[] forwardVariables;
            delete[] backwardVariables;
            return 0.0;
        }

        ctc_vector dEdYTerms;
        dEdYTerms.resize(this->size());
        for (int time=0;time<totalTime;time++)
        {
            thrust::fill(dEdYTerms.begin(),dEdYTerms.end(),0);
            const ctc_vector &fvars=forwardVariables[time];
            const ctc_vector &bvars=backwardVariables[time];
            for (int s=0;s<totalSegments;s++)
            {
                int k=s&1?targetLabelSeq[s/2]:blank;
                dEdYTerms[k]+=fvars[s]*bvars[s];
            }
            for (int i=0;i<this->size();i++)
            {
                internal::Log t=dEdYTerms[i]/(logProb*activation(id,time,i));
                outputErrors[(time*seqNum+id)*this->size()+i]=-(dEdYTerms[i]/(logProb*activation(id,time,i))).exp();
            }
        }

        for (int i=0;i<totalTime;i++)
        {
            forwardVariables[i].clear();
            backwardVariables[i].clear();
        }
        delete[] forwardVariables;
        delete[] backwardVariables;

        return(-logProb.log());
    }

    template <typename TDevice> real_t CtcLayer <TDevice>::activation(int id,int time,int offset)
    {
        return thrust::max(internal::Log::expMin,output[(time*seqNum+id)*this->size()+offset]);
    }

    template <typename TDevice> thrust::pair <int,int> CtcLayer <TDevice>::segment_range(int time) const
    {
        int start=thrust::max(0,totalSegments-(2*(totalTime-time)));
        int end=thrust::min(totalSegments,2*(time+1));
        return(thrust::make_pair(start,end));
    }

    template <typename TDevice> const std::string &CtcLayer <TDevice>::type() const
    {
        static std::string s="connectionist_temporal_classification";
        return(s);
    }

    template <typename TDevice> void CtcLayer <TDevice>::loadSequences(const data_sets::DataSetFraction &fraction)
    {

        PostOutputLayer <TDevice>::loadSequences(fraction);

        seqNum=fraction.targetClasses().size()/fraction.maxSeqLength();
        realSeqNum=fraction.numSequences();
        targetLabel=new int_vector[realSeqNum];

        for (int i=0;i<realSeqNum;i++)
        {
            targetLabel[i].resize(fraction.seqInfo(i).length);
            for (int j=0;j<targetLabel[i].size();j++)
                targetLabel[i][j]=fraction.targetClasses()[j*seqNum+i];
        }
    }

    template <typename TDevice> void CtcLayer <TDevice>::computeForwardPass()
    {
    }

    template <typename TDevice> void CtcLayer <TDevice>::computeBackwardPass()
    {
        thrust::copy(outputErrors.begin(),outputErrors.end(),this->_outputErrors().begin());
    }

    template class CtcLayer <Cpu>;
    template class CtcLayer <Gpu>;

}
